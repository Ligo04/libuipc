#include <finite_element/matrix_utils.h>
#include <muda/ext/eigen/svd.h>
#include <muda/ext/eigen/evd.h>
#include <algorithm/qr_svd.hpp>
namespace uipc::backend::cuda
{
UIPC_GENERIC Vector9 flatten(const Matrix3x3& A) noexcept
{
    Vector9 column;

    unsigned int index = 0;
    for(unsigned int j = 0; j < A.cols(); j++)
        for(unsigned int i = 0; i < A.rows(); i++, index++)
            column[index] = A(i, j);

    return column;
}

UIPC_GENERIC Matrix3x3 unflatten(const Vector9& v) noexcept
{
    Matrix3x3      A;
    unsigned int index = 0;
    for(unsigned int j = 0; j < A.cols(); j++)
        for(unsigned int i = 0; i < A.rows(); i++, index++)
            A(i, j) = v[index];

    return A;
}

UIPC_GENERIC Float ddot(const Matrix3x3& A, const Matrix3x3& B)
{
    Float result = 0;
    for(int y = 0; y < 3; y++)
        for(int x = 0; x < 3; x++)
            result += A(x, y) * B(x, y);

    return result;
}

UIPC_GENERIC void svd(const Matrix3x3& F, Matrix3x3& U, Vector3& Sigma, Matrix3x3& V) noexcept
{
    math::qr_svd(F, Sigma, U, V);
}

UIPC_GENERIC void polar_decomposition(const Matrix3x3& F, Matrix3x3& R, Matrix3x3& S) noexcept
{
    // this function is already tested in the muda eigen test
    muda::eigen::pd(F, R, S);
}

UIPC_GENERIC void evd(const Matrix3x3& A, Vector3& eigen_values, Matrix3x3& eigen_vectors) noexcept
{
    // this function is already tested in the muda eigen test
    muda::eigen::evd(A, eigen_values, eigen_vectors);
}

UIPC_GENERIC void evd(const Matrix9x9& A, Vector9& eigen_values, Matrix9x9& eigen_vectors) noexcept
{
    // this function is already tested in the muda eigen test
    muda::eigen::evd(A, eigen_values, eigen_vectors);
}

UIPC_GENERIC void evd(const Matrix12x12& A, Vector12& eigen_values, Matrix12x12& eigen_vectors) noexcept
{
    muda::eigen::evd(A, eigen_values, eigen_vectors);
}

UIPC_GENERIC Matrix9x9 clamp_to_spd(const Matrix9x9& A) noexcept
{
    // clamp directly
    Matrix9x9 Q;
    Vector9 values;
    muda::eigen::evd(A, values, Q);
    for(int x = 0; x < 9; x++)
        values[x] = (values[x] > 0.0) ? values[x] : 0.0;
    Matrix9x9 B = Q * values.asDiagonal() * Q.transpose();
    return B;
}

UIPC_GENERIC Matrix12x12 clamp_to_spd(const Matrix12x12& A) noexcept
{
    // clamp directly
    Matrix12x12 Q;
    Vector12 values;
    muda::eigen::evd(A, values, Q);
    for(int x = 0; x < 12; x++)
        values[x] = (values[x] > 0.0) ? values[x] : 0.0;
    Matrix12x12 B = Q * values.asDiagonal() * Q.transpose();
    return B;
}
}  // namespace apba
