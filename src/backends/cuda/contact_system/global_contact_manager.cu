#include "hip/hip_runtime.h"
#include <contact_system/global_contact_manager.h>
#include <collision_detection/global_trajectory_filter.h>
#include <sim_engine.h>
#include <contact_system/contact_reporter.h>
#include <contact_system/contact_receiver.h>
#include <uipc/common/enumerate.h>
#include <kernel_cout.h>
#include <uipc/common/unit.h>
#include <uipc/common/zip.h>

namespace uipc::backend
{
template <>
class SimSystemCreator<cuda::GlobalContactManager>
{
  public:
    static U<cuda::GlobalContactManager> create(cuda::SimEngine& engine)
    {
        auto contact_enable_attr =
            engine.world().scene().config().find<IndexT>("contact/enable");
        bool contact_enable = contact_enable_attr->view()[0] != 0;

        auto& types = engine.world().scene().constitution_tabular().types();
        bool  has_inter_primitive_constitution =
            types.find(std::string{builtin::InterPrimitive}) != types.end();

        if(contact_enable || has_inter_primitive_constitution)
            return make_unique<cuda::GlobalContactManager>(engine);
        return nullptr;
    }
};
}  // namespace uipc::backend

namespace uipc::backend::cuda
{
REGISTER_SIM_SYSTEM(GlobalContactManager);

void GlobalContactManager::do_build()
{
    const auto& config = world().scene().config();

    m_impl.global_vertex_manager    = require<GlobalVertexManager>();
    m_impl.global_trajectory_filter = find<GlobalTrajectoryFilter>();


    auto d_hat_attr = config.find<Float>("contact/d_hat");
    m_impl.d_hat    = d_hat_attr->view()[0];

    auto dt_attr = config.find<Float>("dt");
    m_impl.dt    = dt_attr->view()[0];

    auto eps_velocity_attr = config.find<Float>("contact/eps_velocity");
    m_impl.eps_velocity    = eps_velocity_attr->view()[0];

    auto cfl_enable_attr = config.find<IndexT>("cfl/enable");
    m_impl.cfl_enabled   = cfl_enable_attr->view()[0] != 0;

    m_impl.kappa = world().scene().contact_tabular().default_model().resistance();
}

muda::CBuffer2DView<IndexT> GlobalContactManager::contact_mask_tabular() const noexcept
{
    return m_impl.contact_mask_tabular;
}

muda::CBuffer2DView<IndexT> GlobalContactManager::subscene_contact_mask_tabular() const noexcept
{
    return m_impl.contact_mask_tabular_subscene;
}

muda::CBCOOVectorView<Float, 3> GlobalContactManager::contact_gradient() const noexcept
{
    return m_impl.sorted_contact_gradient.view();
}

muda::CBCOOMatrixView<Float, 3> GlobalContactManager::contact_hessian() const noexcept
{
    return m_impl.sorted_contact_hessian.view();
}

void GlobalContactManager::Impl::init(WorldVisitor& world)
{
    // 1) init tabular
    auto contact_models = world.scene().contact_tabular().contact_models();
    auto subscene_contact_models =
        world.scene().contact_tabular().subscene_contact_models();

    auto attr_topo          = contact_models.find<Vector2i>("topo");
    auto attr_resistance    = contact_models.find<Float>("resistance");
    auto attr_friction_rate = contact_models.find<Float>("friction_rate");
    auto attr_enabled       = contact_models.find<IndexT>("is_enabled");

    auto attr_subscene_topo = subscene_contact_models.find<Vector2i>("topo");
    auto attr_subscene_enabled = subscene_contact_models.find<IndexT>("is_enabled");

    UIPC_ASSERT(attr_topo != nullptr, "topo is not found in contact tabular");
    UIPC_ASSERT(attr_resistance != nullptr, "resistance is not found in contact tabular");
    UIPC_ASSERT(attr_friction_rate != nullptr, "friction_rate is not found in contact tabular");
    UIPC_ASSERT(attr_enabled != nullptr, "is_enabled is not found in contact tabular");

    UIPC_ASSERT(attr_subscene_topo != nullptr, "subscene topo is not found in contact tabular");
    UIPC_ASSERT(attr_subscene_enabled != nullptr,
                "subscene is_enabled is not found in contact tabular");

    auto topo_view            = attr_topo->view();
    auto resistance_view      = attr_resistance->view();
    auto friction_rate_view   = attr_friction_rate->view();
    auto enabled_view         = attr_enabled->view();
    auto subscene_topo_view   = attr_subscene_topo->view();
    auto subscene_enable_view = attr_subscene_enabled->view();

    auto N  = world.scene().contact_tabular().element_count();
    auto SN = world.scene().contact_tabular().subscene_element_count();

    h_contact_tabular.resize(
        N * N, ContactCoeff{.kappa = resistance_view[0], .mu = friction_rate_view[0]});

    h_contact_mask_tabular.resize(N * N, 1);
    h_contact_mask_tabular_subscene.resize(SN * SN, 1);

    for(auto&& [ids, kappa, mu, is_enabled] :
        zip(topo_view, resistance_view, friction_rate_view, enabled_view))
    {

        ContactCoeff coeff{.kappa = kappa, .mu = mu};

        auto upper                    = ids.x() * N + ids.y();
        h_contact_tabular[upper]      = coeff;
        h_contact_mask_tabular[upper] = is_enabled;

        auto lower                    = ids.y() * N + ids.x();
        h_contact_tabular[lower]      = coeff;
        h_contact_mask_tabular[lower] = is_enabled;
    }

    for(auto&& [ids, is_enabled] : zip(subscene_topo_view, subscene_enable_view))
    {
        auto upper                             = ids.x() * SN + ids.y();
        h_contact_mask_tabular_subscene[upper] = is_enabled;

        auto lower                             = ids.y() * SN + ids.x();
        h_contact_mask_tabular_subscene[lower] = is_enabled;
    }

    contact_tabular.resize(muda::Extent2D{N, N});
    contact_tabular.view().copy_from(h_contact_tabular.data());

    contact_mask_tabular.resize(muda::Extent2D{N, N});
    contact_mask_tabular.view().copy_from(h_contact_mask_tabular.data());

    contact_mask_tabular_subscene.resize(muda::Extent2D{SN, SN});
    contact_mask_tabular_subscene.view().copy_from(
        h_contact_mask_tabular_subscene.data());

    // 2) vertex contact info
    vert_is_active_contact.resize(global_vertex_manager->positions().size(), 0);
    vert_disp_norms.resize(global_vertex_manager->positions().size(), 0.0);

    // 3) reporters
    auto contact_reporter_view = contact_reporters.view();
    for(auto&& [i, R] : enumerate(contact_reporter_view))
        R->init();
    for(auto&& [i, R] : enumerate(contact_reporter_view))
        R->m_index = i;

    reporter_energy_offsets_counts.resize(contact_reporter_view.size());
    reporter_gradient_offsets_counts.resize(contact_reporter_view.size());
    reporter_hessian_offsets_counts.resize(contact_reporter_view.size());

    // 4) receivers
    auto contact_receiver_view = contact_receivers.view();
    for(auto&& [i, R] : enumerate(contact_receiver_view))
        R->init();
    for(auto&& [i, R] : enumerate(contact_receiver_view))
        R->m_index = i;

    classified_contact_gradients.resize(contact_receiver_view.size());
    classified_contact_hessians.resize(contact_receiver_view.size());
}

void GlobalContactManager::Impl::compute_d_hat()
{
    // TODO: Now do nothing
}

void GlobalContactManager::Impl::compute_adaptive_kappa()
{
    // TODO: Now do nothing
}

Float GlobalContactManager::Impl::compute_cfl_condition()
{
    if(!cfl_enabled)  // if cfl is disabled, just return 1.0
        return 1.0;

    vert_is_active_contact.fill(0);  // clear the active flag

    if(global_trajectory_filter)
    {
        global_trajectory_filter->label_active_vertices();

        auto displacements = global_vertex_manager->displacements();

        using namespace muda;
        ParallelFor()
            .file_line(__FILE__, __LINE__)
            .apply(displacements.size(),
                   [disps      = displacements.cviewer().name("disp"),
                    disp_norms = vert_disp_norms.viewer().name("disp_norm"),
                    is_contact_active = vert_is_active_contact.viewer().name(
                        "vert_is_contact_active")] __device__(int i) mutable
                   {
                       // if the contact is not active, then the displacement is ignored
                       disp_norms(i) = is_contact_active(i) ? disps(i).norm() : 0.0;
                   });

        DeviceReduce().Max(vert_disp_norms.data(),
                           max_disp_norm.data(),
                           vert_disp_norms.size());

        Float h_max_disp_norm = max_disp_norm;
        return h_max_disp_norm == 0.0 ? 1.0 : std::min(0.5 * d_hat / h_max_disp_norm, 1.0);
    }
    else
    {
        return 1.0;
    }
}

void GlobalContactManager::Impl::compute_contact()
{
    _assemble();
    _convert_matrix();
    _distribute();
}

void GlobalContactManager::Impl::_assemble()
{
    auto vertex_count = global_vertex_manager->positions().size();

    auto reporter_gradient_counts = reporter_gradient_offsets_counts.counts();
    auto reporter_hessian_counts  = reporter_hessian_offsets_counts.counts();

    for(auto&& [i, reporter] : enumerate(contact_reporters.view()))
    {
        GradientHessianExtentInfo info;
        reporter->report_gradient_hessian_extent(info);
        reporter_gradient_counts[i] = info.m_gradient_count;
        reporter_hessian_counts[i]  = info.m_hessian_count;
        spdlog::info("<{}> contact Grad3 count: {}, contact Hess3x3 count: {}",
                     reporter->name(),
                     info.m_gradient_count,
                     info.m_hessian_count);
    }

    // scan
    reporter_gradient_offsets_counts.scan();
    reporter_hessian_offsets_counts.scan();

    auto total_gradient_count = reporter_gradient_offsets_counts.total_count();
    auto total_hessian_count  = reporter_hessian_offsets_counts.total_count();

    // allocate
    loose_resize_entries(collected_contact_gradient, total_gradient_count);
    loose_resize_entries(sorted_contact_gradient, total_gradient_count);
    loose_resize_entries(collected_contact_hessian, total_hessian_count);
    loose_resize_entries(sorted_contact_hessian, total_hessian_count);
    collected_contact_gradient.reshape(vertex_count);
    collected_contact_hessian.reshape(vertex_count, vertex_count);

    // collect
    for(auto&& [i, reporter] : enumerate(contact_reporters.view()))
    {
        auto [g_offset, g_count] = reporter_gradient_offsets_counts[i];
        auto [h_offset, h_count] = reporter_hessian_offsets_counts[i];


        GradientHessianInfo info;

        info.m_gradient = collected_contact_gradient.view().subview(g_offset, g_count);
        info.m_hessian = collected_contact_hessian.view().subview(h_offset, h_count);

        reporter->assemble(info);
    }
}

void GlobalContactManager::Impl::_convert_matrix()
{
    matrix_converter.convert(collected_contact_hessian, sorted_contact_hessian);
    matrix_converter.convert(collected_contact_gradient, sorted_contact_gradient);
}

void GlobalContactManager::Impl::_distribute()
{
    using namespace muda;

    auto vertex_count = global_vertex_manager->positions().size();

    for(auto&& [i, receiver] : enumerate(contact_receivers.view()))
    {
        ClassifyInfo info;
        receiver->report(info);

        auto& classified_gradients = classified_contact_gradients[i];
        classified_gradients.reshape(vertex_count);
        auto& classified_hessians = classified_contact_hessians[i];
        classified_hessians.reshape(vertex_count, vertex_count);

        // 1) report gradient
        if(info.is_diag())
        {
            const auto N = sorted_contact_gradient.doublet_count();

            // clear the range in device
            gradient_range = Vector2i{0, 0};

            // partition
            ParallelFor()
                .kernel_name(__FUNCTION__)
                .apply(N,
                       [gradient_range = gradient_range.viewer().name("gradient_range"),
                        contact_gradient =
                            std::as_const(sorted_contact_gradient).viewer().name("contact_gradient"),
                        range = info.m_gradient_i_range] __device__(int I) mutable
                       {
                           auto in_range = [](int i, const Vector2i& range)
                           { return i >= range.x() && i < range.y(); };

                           auto&& [i, G]      = contact_gradient(I);
                           bool this_in_range = in_range(i, range);

                           //cout << "I: " << I << ", i: " << i << ", G: " << G
                           //     << ", in_range: " << this_in_range << "\n";

                           if(!this_in_range)
                           {
                               return;
                           }

                           bool prev_in_range = false;
                           if(I > 0)
                           {
                               auto&& [prev_i, prev_G] = contact_gradient(I - 1);
                               prev_in_range = in_range(prev_i, range);
                           }
                           bool next_in_range = false;
                           if(I < contact_gradient.total_doublet_count() - 1)
                           {
                               auto&& [next_i, next_G] = contact_gradient(I + 1);
                               next_in_range = in_range(next_i, range);
                           }

                           // if the prev is not in range, then this is the start of the partition
                           if(!prev_in_range)
                           {
                               gradient_range->x() = I;
                           }
                           // if the next is not in range, then this is the end of the partition
                           if(!next_in_range)
                           {
                               gradient_range->y() = I + 1;
                           }
                       });

            Vector2i h_range = gradient_range;  // copy back

            auto count = h_range.y() - h_range.x();

            loose_resize_entries(classified_gradients, count);

            // fill
            if(count > 0)
            {
                ParallelFor()
                    .kernel_name(__FUNCTION__)
                    .apply(count,
                           [contact_gradient =
                                std::as_const(sorted_contact_gradient).viewer().name("contact_gradient"),
                            classified_gradient = classified_gradients.viewer().name("classified_gradient"),
                            range = h_range] __device__(int I) mutable
                           {
                               auto&& [i, G] = contact_gradient(range.x() + I);
                               classified_gradient(I).write(i, G);
                           });
            }
        }

        // 2) report hessian
        if(!info.is_empty())
        {
            const auto N = sorted_contact_hessian.triplet_count();

            // +1 for calculate the total count
            loose_resize(selected_hessian, N + 1);
            loose_resize(selected_hessian_offsets, N + 1);

            // select
            ParallelFor()
                .kernel_name(__FUNCTION__)
                .apply(
                    N,
                    [selected_hessian = selected_hessian.view(0, N).viewer().name("selected_hessian"),
                     last =
                         VarView<IndexT>{selected_hessian.data() + N}.viewer().name("last"),
                     contact_hessian = sorted_contact_hessian.cviewer().name("contact_hessian"),
                     i_range = info.m_hessian_i_range,
                     j_range = info.m_hessian_j_range] __device__(int I) mutable
                    {
                        auto&& [i, j, H] = contact_hessian(I);

                        auto in_range = [](int i, const Vector2i& range)
                        { return i >= range.x() && i < range.y(); };

                        selected_hessian(I) =
                            in_range(i, i_range) && in_range(j, j_range) ? 1 : 0;

                        // fill the last one as 0, so that we can calculate the total count
                        // during the exclusive scan
                        if(I == 0)
                            last = 0;
                    });

            // scan
            DeviceScan().ExclusiveSum(selected_hessian.data(),
                                      selected_hessian_offsets.data(),
                                      selected_hessian.size());

            IndexT h_total_count = 0;
            VarView<IndexT>{selected_hessian_offsets.data() + N}.copy_to(&h_total_count);

            loose_resize_entries(classified_hessians, h_total_count);

            // fill
            if(h_total_count > 0)
            {
                ParallelFor()
                    .kernel_name(__FUNCTION__)
                    .apply(N,
                           [selected_hessian = selected_hessian.cviewer().name("selected_hessian"),
                            selected_hessian_offsets =
                                selected_hessian_offsets.cviewer().name("selected_hessian_offsets"),
                            contact_hessian = sorted_contact_hessian.cviewer().name("contact_hessian"),
                            classified_hessian = classified_hessians.viewer().name("classified_hessian"),
                            i_range = info.m_hessian_i_range,
                            j_range = info.m_hessian_j_range] __device__(int I) mutable
                           {
                               if(selected_hessian(I))
                               {
                                   auto&& [i, j, H] = contact_hessian(I);
                                   auto offset = selected_hessian_offsets(I);

                                   classified_hessian(offset).write(i, j, H);
                               }
                           });
            }

            ClassifiedContactInfo classified_info;

            classified_info.m_gradient = classified_gradients.view();
            classified_info.m_hessian  = classified_hessians.view();

            receiver->receive(classified_info);
        }
    }
}

void GlobalContactManager::Impl::loose_resize_entries(muda::DeviceTripletMatrix<Float, 3>& m,
                                                      SizeT size)
{
    if(size > m.triplet_capacity())
    {
        m.reserve_triplets(size * reserve_ratio);
    }
    m.resize_triplets(size);
}

void GlobalContactManager::Impl::loose_resize_entries(muda::DeviceDoubletVector<Float, 3>& v,
                                                      SizeT size)
{
    if(size > v.doublet_capacity())
    {
        v.reserve_doublets(size * reserve_ratio);
    }
    v.resize_doublets(size);
}


void GlobalContactManager::ClassifyInfo::range(const Vector2i& LRange, const Vector2i& RRange)
{
    m_type             = Type::Range;
    m_hessian_i_range  = LRange;
    m_hessian_j_range  = RRange;
    m_gradient_i_range = Vector2i::Zero();
}

void GlobalContactManager::ClassifyInfo::range(const Vector2i& Range)
{
    m_type             = Type::Range;
    m_gradient_i_range = Range;
    m_hessian_i_range  = Range;
    m_hessian_j_range  = Range;
}

bool GlobalContactManager::ClassifyInfo::is_empty() const
{
    return m_hessian_i_range[0] == m_hessian_i_range[1]
           || m_hessian_j_range[0] == m_hessian_j_range[1];
}

bool GlobalContactManager::ClassifyInfo::is_diag() const
{
    return m_gradient_i_range[0] != m_gradient_i_range[1];
}

void GlobalContactManager::ClassifyInfo::sanity_check()
{
    if(is_diag())
    {
        UIPC_ASSERT(m_gradient_i_range.x() <= m_gradient_i_range.y(),
                    "Diagonal Contact Gradient Range is invalid, [{}, {})",
                    m_gradient_i_range.x(),
                    m_gradient_i_range.y());

        UIPC_ASSERT(m_hessian_i_range == m_hessian_j_range,
                    "Diagonal Contact Hessian must have the same i_range and j_range");
    }
    else
    {
        UIPC_ASSERT(m_gradient_i_range.x() == m_gradient_i_range.y(),
                    "Off-Diagonal Contact must not have Gradient Part");
    }

    UIPC_ASSERT(m_hessian_i_range.x() <= m_hessian_i_range.y(),
                "Contact Hessian Range-i is invalid");
    UIPC_ASSERT(m_hessian_j_range.x() <= m_hessian_j_range.y(),
                "Contact Hessian Range-j is invalid");
}
}  // namespace uipc::backend::cuda


namespace uipc::backend::cuda
{
void GlobalContactManager::compute_d_hat()
{
    m_impl.compute_d_hat();
}

void GlobalContactManager::compute_contact()
{
    m_impl.compute_contact();
}

void GlobalContactManager::compute_adaptive_kappa()
{
    m_impl.compute_adaptive_kappa();
}

Float GlobalContactManager::compute_cfl_condition()
{
    return m_impl.compute_cfl_condition();
}

void GlobalContactManager::init()
{
    m_impl.init(world());
}

Float GlobalContactManager::d_hat() const
{
    return m_impl.d_hat;
}
Float GlobalContactManager::eps_velocity() const
{
    return m_impl.eps_velocity;
}
bool GlobalContactManager::cfl_enabled() const
{
    return m_impl.cfl_enabled;
}
void GlobalContactManager::add_reporter(ContactReporter* reporter)
{
    check_state(SimEngineState::BuildSystems, "add_reporter()");
    UIPC_ASSERT(reporter != nullptr, "reporter is nullptr");
    m_impl.contact_reporters.register_subsystem(*reporter);
}
void GlobalContactManager::add_receiver(ContactReceiver* receiver)
{
    check_state(SimEngineState::BuildSystems, "add_receiver()");
    UIPC_ASSERT(receiver != nullptr, "receiver is nullptr");
    m_impl.contact_receivers.register_subsystem(*receiver);
}
muda::CBuffer2DView<ContactCoeff> GlobalContactManager::contact_tabular() const noexcept
{
    return m_impl.contact_tabular;
}
}  // namespace uipc::backend::cuda